#include<iostream>
#include<cstdio>
#include<cstdlib>
#include<hip/hip_runtime.h>
using namespace std;


__global__ void minimum(int *input)
{
	int tid=threadIdx.x;
	auto step_size=1;
  int number_of_threads=blockDim.x;
  
  while(number_of_threads>0)
  {
      if(tid<number_of_threads)
      {
          int first=tid*step_size*2;
          int second=first+step_size;
          if(input[second]<input[first])
            input[first]=input[second];
      }
      step_size=step_size*2;
      number_of_threads/=2;
  }
}

__global__ void max(int *input)
{
   int tid=threadIdx.x;
   auto step_size=1;
   int number_of_threads=blockDim.x;
   while(number_of_threads>0)
   {
       if(tid<number_of_threads)
       {
           int first=tid*step_size*2;
           int second=first+step_size;
           if(input[second]>input[first])
            input[first]=input[second];
       }
       step_size*=2;
       number_of_threads/=2;
   }
}

__global__ void sum(int *input)
{
    const int tid=threadIdx.x;
    auto  step_size=1;
    int number_of_threads=blockDim.x;
    while(number_of_threads>0)
    {
        if(tid<number_of_threads)
        {
            const int first=tid*step_size*2;
            const int second=first+step_size;
            input[first]=input[first]+input[second];
        }
    step_size = step_size*2;; 
		number_of_threads =number_of_threads/2;
    }
}

__global__ void average(int *input) //You can use above sum() to calculate sum and divide it by num_of_elememts
{
    const int tid=threadIdx.x;
    auto  step_size=1;
    int number_of_threads=blockDim.x;
    int totalElements=number_of_threads*2;
    while(number_of_threads>0)
    {
        if(tid<number_of_threads)
        {
            const int first=tid*step_size*2;
            const int second=first+step_size;
            input[first]=input[first]+input[second];
        }
        step_size = step_size*2;; 
		number_of_threads =number_of_threads/2;
    }
    input[0]=input[0]/totalElements;
}

int main()
{
int n;
n=200;
srand(n);
int *arr=new int[n];
int min=20000; //Any Large Number would work
 cout<<"Elements are: "<<endl;
   //# Generate Input array using rand()
	for(int i=0;i<n;i++)
	{
	arr[i]=rand()%n;
      if(arr[i]<min)
        min=arr[i];
    cout<<arr[i]<<" ";
	}
	int size=n*sizeof(int); //calculate no. of bytes for array
	int *arr_d,result1;

//# Allocate memory for min Operation
hipMalloc(&arr_d,size);
hipMemcpy(arr_d,arr,size,hipMemcpyHostToDevice);
minimum<<<1,n/2>>>(arr_d);
hipMemcpy(&result1,arr_d,sizeof(int),hipMemcpyDeviceToHost);
cout<<endl<<"The minimum element is "<<result1<<endl;  
cout<<"The min element (using CPU) is "<<min<<endl;
   
    
//#MAX OPERATION 
int *arr_max,maxValue;
hipMalloc(&arr_max,size);
hipMemcpy(arr_max,arr,size,hipMemcpyHostToDevice);
max<<<1,n/2>>>(arr_max);
hipMemcpy(&maxValue,arr_max,sizeof(int),hipMemcpyDeviceToHost);
cout<<"The maximum element is "<<maxValue<<endl;
    
//SUM OPERATION 
int *arr_sum,sumValue;
hipMalloc(&arr_sum,size);
hipMemcpy(arr_sum,arr,size,hipMemcpyHostToDevice);
sum<<<1,n/2>>>(arr_sum);
hipMemcpy(&sumValue,arr_sum,sizeof(int),hipMemcpyDeviceToHost);
cout<<"The sum of elements is "<<sumValue<<endl; 
cout<<"The average of elements is (by CPU): "<<(sumValue/n)<<endl; 
   
//#AVG OPERATION 
int *arr_avg,avgValue;
hipMalloc(&arr_avg,size);
hipMemcpy(arr_avg,arr,size,hipMemcpyHostToDevice);
average<<<1,n/2>>>(arr_avg);
hipMemcpy(&avgValue,arr_avg,sizeof(int),hipMemcpyDeviceToHost);
cout<<"The average of elements is (by GPU): "<<avgValue<<endl; 
//# Free all allocated device memeory
hipFree(arr_d);
hipFree(arr_sum);
hipFree(arr_max);
hipFree(arr_avg);
return 0;
}